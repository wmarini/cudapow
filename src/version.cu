
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

namespace app {

__host__ int Version()
{
    int n_devices;
    hipError_t err = hipGetDeviceCount(&n_devices);
    if (err != hipSuccess) {
        std::puts(hipGetErrorString(err));
        std::exit(EXIT_FAILURE);
    }
    return n_devices;
}

} // namespace app
